#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <dirent.h>
#include <fstream>
#include <iostream>
#include <stdint.h>
#include <sys/stat.h>
#include <thrust/copy.h>

#include "../include/kernel/lorenzo_var.cuh"
#include "../include/utils/cuda_err.cuh"

#define UINT32_BIT_LEN 32
// #define VERIFICATION
// #define DEBUG
__global__ void printFirst(float *dev_a)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
        printf("float First element: %f\n", dev_a[0]);
}

__global__ void printFirst_uint8(uint8_t *dev_a)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
        printf("uint8 First element: %d\n", dev_a[0]);
}

long GetFileSize(std::string fidataTypeLename)
{
    struct stat stat_buf;
    int rc = stat(fidataTypeLename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : -1;
}

template <typename T>
T *read_binary_to_new_array(const std::string &fname, size_t dtype_dataTypeLen)
{
    std::ifstream ifs(fname.c_str(), std::ios::binary | std::ios::in);
    if (not ifs.is_open())
    {
        std::cerr << "fail to open " << fname << std::endl;
        exit(1);
    }
    auto _a = new T[dtype_dataTypeLen]();
    ifs.read(reinterpret_cast<char *>(_a), std::streamsize(dtype_dataTypeLen * sizeof(T)));
    ifs.close();
    return _a;
}

template <typename T>
void write_array_to_binary(const std::string &fname, T *const _a, size_t const dtype_dataTypeLen)
{
    std::ofstream ofs(fname.c_str(), std::ios::binary | std::ios::out);
    if (not ofs.is_open())
        return;
    ofs.write(reinterpret_cast<const char *>(_a), std::streamsize(dtype_dataTypeLen * sizeof(T)));
    ofs.close();
}

__global__ void compressionFusedKernel(
    const uint32_t *__restrict__ in,
    uint32_t *out,
    uint32_t *deviceOffsetCounter,
    uint32_t *deviceBitFlagArr,
    uint32_t *deviceStartPosition,
    uint32_t *deviceCompressedSize)
{
    // 32 x 32 data chunk size with one padding for each row, overall 4096 bytes per chunk
    __shared__ uint32_t dataChunk[32][33];
    __shared__ uint16_t byteFlagArray[257];
    __shared__ uint32_t bitflagArr[8];
    __shared__ uint32_t startPosition;

    uint32_t byteFlag = 0;
    uint32_t v;

    v = in[threadIdx.x + threadIdx.y * 32 + blockIdx.x * 1024];
    __syncthreads();

#ifdef DEBUG
    dataChunk[threadIdx.y][threadIdx.x] = v;
    if (threadIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("original data:\n");
        for (int tmpIdx = 0; tmpIdx < 32; tmpIdx++)
        {
            printf("%u\t", dataChunk[0][tmpIdx]);
        }
        printf("\n");
    }
#endif

#pragma unroll 32
    for (int i = 0; i < 32; i++)
    {
        dataChunk[threadIdx.y][i] = __ballot_sync(0xFFFFFFFFU, v & (1U << i));
    }
    __syncthreads();

#ifdef DEBUG
    if (threadIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("shuffled data:\n");
        for (int tmpIdx = 0; tmpIdx < 32; tmpIdx++)
        {
            printf("%u\t", dataChunk[0][tmpIdx]);
        }
        printf("\n");
    }
#endif

    // generate byteFlagArray
    if (threadIdx.x < 8)
    {
#pragma unroll 4
        for (int i = 0; i < 4; i++)
        {
            byteFlag |= dataChunk[threadIdx.x * 4 + i][threadIdx.y];
        }
        byteFlagArray[threadIdx.y * 8 + threadIdx.x] = byteFlag > 0;
    }
    __syncthreads();

    // generate bitFlagArray
    uint32_t buffer;
    if (threadIdx.y < 8)
    {
        buffer = byteFlagArray[threadIdx.y * 32 + threadIdx.x];
        bitflagArr[threadIdx.y] = __ballot_sync(0xFFFFFFFFU, buffer);
    }
    __syncthreads();

#ifdef DEBUG
    if (threadIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("bit flag array: %u\n", bitflagArr[0]);
    }
#endif

    // write back bitFlagArray to global memory
    if (threadIdx.x < 8 && threadIdx.y == 0)
    {
        deviceBitFlagArr[blockIdx.x * 8 + threadIdx.x] = bitflagArr[threadIdx.x];
    }

    int blockSize = 256;
    int tid = threadIdx.x + threadIdx.y * 32;

    // prefix summation, up-sweep
    int prefixSumOffset = 1;
#pragma unroll 8
    for (int d = 256 >> 1; d > 0; d = d >> 1)
    {
        if (tid < d)
        {
            int ai = prefixSumOffset * (2 * tid + 1) - 1;
            int bi = prefixSumOffset * (2 * tid + 2) - 1;
            byteFlagArray[bi] += byteFlagArray[ai];
        }
        __syncthreads();
        prefixSumOffset *= 2;
    }

    // clear the last element
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        byteFlagArray[blockSize] = byteFlagArray[blockSize - 1];
        byteFlagArray[blockSize - 1] = 0;
    }
    __syncthreads();

// prefix summation, down-sweep
#pragma unroll 8
    for (int d = 1; d < 256; d *= 2)
    {
        prefixSumOffset >>= 1;
        if (tid < d)
        {
            int ai = prefixSumOffset * (2 * tid + 1) - 1;
            int bi = prefixSumOffset * (2 * tid + 2) - 1;

            uint32_t t = byteFlagArray[ai];
            byteFlagArray[ai] = byteFlagArray[bi];
            byteFlagArray[bi] += t;
        }
        __syncthreads();
    }

#ifdef DEBUG
    if (threadIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("byte flag array:\n");
        for (int tmpIdx = 0; tmpIdx < 32; tmpIdx++)
        {
            printf("%u\t", byteFlagArray[tmpIdx]);
        }
        printf("\n");
    }
#endif

    // use atomicAdd to reserve a space for compressed data chunk
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        startPosition = atomicAdd(deviceOffsetCounter, byteFlagArray[blockSize] * 4);
        deviceStartPosition[blockIdx.x] = startPosition;
        deviceCompressedSize[blockIdx.x] = byteFlagArray[blockSize];
    }
    __syncthreads();

    // write back the compressed data based on the startPosition
    int flagIndex = floorf(tid / 4);
    if (byteFlagArray[flagIndex + 1] != byteFlagArray[flagIndex])
    {
        out[startPosition + byteFlagArray[flagIndex] * 4 + tid % 4] = dataChunk[threadIdx.x][threadIdx.y];
    }
}

__global__ void decompressionFusedKernel(
    uint32_t *deviceInput,
    uint32_t *deviceOutput,
    uint32_t *deviceBitFlagArr,
    uint32_t *deviceStartPosition)
{
    // allocate shared byte flag array
    __shared__ uint32_t dataChunk[32][33];
    __shared__ uint16_t byteFlagArray[257];
    __shared__ uint32_t startPosition;

    // there are 32 x 32 uint32_t in this data chunk
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int bid = blockIdx.x;

    // transfer bit flag array to byte flag array
    uint32_t bitFlag = 0;
    if (threadIdx.x < 8 && threadIdx.y == 0)
    {
        bitFlag = deviceBitFlagArr[bid * 8 + threadIdx.x];
#pragma unroll 32
        for (int tmpInd = 0; tmpInd < 32; tmpInd++)
        {
            byteFlagArray[threadIdx.x * 32 + tmpInd] = (bitFlag & (1U << tmpInd)) > 0;
        }
    }
    __syncthreads();

    int prefixSumOffset = 1;
    int blockSize = 256;

// prefix summation, up-sweep
#pragma unroll 8
    for (int d = 256 >> 1; d > 0; d = d >> 1)
    {
        if (tid < d)
        {
            int ai = prefixSumOffset * (2 * tid + 1) - 1;
            int bi = prefixSumOffset * (2 * tid + 2) - 1;
            byteFlagArray[bi] += byteFlagArray[ai];
        }
        __syncthreads();
        prefixSumOffset *= 2;
    }

    // clear the last element
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        byteFlagArray[blockSize] = byteFlagArray[blockSize - 1];
        byteFlagArray[blockSize - 1] = 0;
    }
    __syncthreads();

// prefix summation, down-sweep
#pragma unroll 8
    for (int d = 1; d < 256; d *= 2)
    {
        prefixSumOffset >>= 1;
        if (tid < d)
        {
            int ai = prefixSumOffset * (2 * tid + 1) - 1;
            int bi = prefixSumOffset * (2 * tid + 2) - 1;

            uint32_t t = byteFlagArray[ai];
            byteFlagArray[ai] = byteFlagArray[bi];
            byteFlagArray[bi] += t;
        }
        __syncthreads();
    }

#ifdef DEBUG
    if (threadIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("decompressed byte flag array:\n");
        for (int tmpIdx = 0; tmpIdx < 32; tmpIdx++)
        {
            printf("%u\t", byteFlagArray[tmpIdx]);
        }
        printf("\n");
    }
#endif

    // initialize the shared memory to all 0
    dataChunk[threadIdx.y][threadIdx.x] = 0;
    __syncthreads();

    // get the start position
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        startPosition = deviceStartPosition[bid];
    }
    __syncthreads();

    // write back shuffled data to shared mem
    int byteFlagInd = tid / 4;
    if (byteFlagArray[byteFlagInd + 1] != byteFlagArray[byteFlagInd])
    {
        dataChunk[threadIdx.x][threadIdx.y] = deviceInput[startPosition + byteFlagArray[byteFlagInd] * 4 + tid % 4];
    }
    __syncthreads();

    // store the corresponding uint32 to the register buffer
    uint32_t buffer = dataChunk[threadIdx.y][threadIdx.x];
    __syncthreads();

// bitshuffle (reverse)
#pragma unroll 32
    for (int i = 0; i < 32; i++)
    {
        dataChunk[threadIdx.y][i] = __ballot_sync(0xFFFFFFFFU, buffer & (1U << i));
    }
    __syncthreads();

#ifdef DEBUG
    if (threadIdx.y == 0 && threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("decomopressed data:\n");
        for (int tmpIdx = 0; tmpIdx < 32; tmpIdx++)
        {
            printf("%u\t", dataChunk[0][tmpIdx]);
        }
        printf("\n");
    }
#endif

    // write back to global memory
    deviceOutput[tid + bid * blockDim.x * blockDim.y] = dataChunk[threadIdx.y][threadIdx.x];
}

void fzCompress(float *deviceInput, uint8_t *deviceCompressed, int *outputSizePtr, int inputSize,
                int x, int y, int z, float eb, uint16_t *quantizationCode)
{
    printFirst<<<1, 1>>>(deviceInput);
    printFirst_uint8<<<1, 1>>>(deviceCompressed);
    std::cout << "input size" << inputSize << std::endl
              << "x" << x << std::endl
              << "y" << y << std::endl
              << "z" << z << std::endl;
    // defination of some basic variables
    auto inputDimension = dim3(x, y, z);
    auto dataTypeLen = inputSize;
    float timeElapsed;
    uint32_t offsetSum;

    // defination of device pointers
    uint16_t *deviceCompressedOutput;
    uint32_t *deviceBitFlagArr;
    uint32_t *deviceStartPosition;
    uint8_t *deviceCompressedStartPosition;
    deviceCompressedStartPosition = deviceCompressed + sizeof(int) * 5;

    bool *deviceSignNum;
    uint16_t *deviceQuantizationCode = quantizationCode;
    uint32_t *deviceOffsetCounter;
    uint32_t *deviceCompressedSize;

    // defination of timers
    std::chrono::time_point<std::chrono::system_clock> compressionStart, compressionEnd;

    // to calculate some usefule constants
    int blockSize = 16;
    auto quantizationCodeByteLen = dataTypeLen * 2; // quantization code length in unit of bytes
    quantizationCodeByteLen = quantizationCodeByteLen % 4096 == 0 ? quantizationCodeByteLen : quantizationCodeByteLen - quantizationCodeByteLen % 4096 + 4096;
    auto paddingDataTypeLen = quantizationCodeByteLen / 2;
    int bitFlagArrSize = quantizationCodeByteLen % (blockSize * UINT32_BIT_LEN) == 0 ? quantizationCodeByteLen / (blockSize * UINT32_BIT_LEN) : int(quantizationCodeByteLen / (blockSize * UINT32_BIT_LEN)) + 1;

    dim3 block(32, 32);
    dim3 grid(floor(paddingDataTypeLen / 2048)); // divided by 2 is because the file is transformed from uint32 to uint16

    CHECK_CUDA(hipMalloc((void **)&deviceQuantizationCode, sizeof(uint16_t) * paddingDataTypeLen));
    CHECK_CUDA(hipMemset(deviceQuantizationCode, 0, sizeof(uint16_t) * paddingDataTypeLen));
    CHECK_CUDA(hipMalloc((void **)&deviceSignNum, sizeof(bool) * paddingDataTypeLen));

    CHECK_CUDA(hipMalloc((void **)&deviceOffsetCounter, sizeof(uint32_t)));
    CHECK_CUDA(hipMalloc((void **)&deviceCompressedSize, sizeof(uint32_t) * floor(quantizationCodeByteLen / 4096)));

    // cuda copy some info to compressed data
    CHECK_CUDA(hipMemcpy(deviceCompressed, &inputSize, sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(deviceCompressed + sizeof(int), &x, sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(deviceCompressed + sizeof(int) * 2, &y, sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(deviceCompressed + sizeof(int) * 3, &z, sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(deviceCompressed + sizeof(int) * 4, &eb, sizeof(float), hipMemcpyHostToDevice));

    // get the differents by the calculated offset
    int offsetCalculator = 0;
    deviceBitFlagArr = (uint32_t *)(deviceCompressedStartPosition + offsetCalculator);
    offsetCalculator += sizeof(uint32_t) * bitFlagArrSize;
    deviceStartPosition = (uint32_t *)(deviceCompressedStartPosition + offsetCalculator);
    offsetCalculator += sizeof(uint32_t) * floor(quantizationCodeByteLen / 4096);
    deviceCompressedOutput = (uint16_t *)(deviceCompressedStartPosition + offsetCalculator);

    hipStream_t stream;
    hipStreamCreate(&stream);

    compressionStart = std::chrono::system_clock::now();

    // pre-quantization
    cusz::experimental::launch_construct_LorenzoI_var<float, uint16_t, float>(deviceInput, deviceQuantizationCode, deviceSignNum, inputDimension, eb, timeElapsed, stream);

    // bitshuffle kernel
    compressionFusedKernel<<<grid, block>>>((uint32_t *)deviceQuantizationCode, (uint32_t *)deviceCompressedOutput, deviceOffsetCounter, deviceBitFlagArr, deviceStartPosition, deviceCompressedSize);

    hipDeviceSynchronize();
    printFirst<<<1, 1>>>(deviceInput);
    printFirst_uint8<<<1, 1>>>(deviceCompressed);
    compressionEnd = std::chrono::system_clock::now();

    CHECK_CUDA(hipMemcpy(&offsetSum, deviceOffsetCounter, sizeof(uint32_t), hipMemcpyDeviceToHost));
    printf("original size: %d\n", sizeof(float) * inputSize);
    printf("compressed size: %ld\n", sizeof(uint32_t) * bitFlagArrSize + offsetSum * sizeof(uint32_t) + sizeof(uint32_t) * int(quantizationCodeByteLen / 4096));
    printf("test1: %ld\n", sizeof(uint32_t) * bitFlagArrSize);
    printf("test2: %ld\n", offsetSum * sizeof(uint32_t));
    printf("test3: %ld\n", sizeof(uint32_t) * int(quantizationCodeByteLen / 4096));

    printf("compression ratio: %f\n", float(inputSize * sizeof(float)) / float(sizeof(uint32_t) * bitFlagArrSize + offsetSum * sizeof(uint32_t) + sizeof(uint32_t) * floor(quantizationCodeByteLen / 4096)));
    *outputSizePtr = sizeof(uint32_t) * bitFlagArrSize + offsetSum * sizeof(uint32_t) + sizeof(uint32_t) * int(quantizationCodeByteLen / 4096) + 5 * sizeof(int);

    std::chrono::duration<double> compressionTime = compressionEnd - compressionStart;

    std::cout << "compression e2e time: " << compressionTime.count() << " s\n";
    std::cout << "compression e2e throughput: " << float(inputSize * sizeof(float)) / 1024 / 1024 / 1024 / compressionTime.count() << " GB/s\n";

    // CHECK_CUDA(hipFree(deviceQuantizationCode));
    CHECK_CUDA(hipFree(deviceSignNum));

    CHECK_CUDA(hipFree(deviceOffsetCounter));
    CHECK_CUDA(hipFree(deviceCompressedSize));

    hipStreamDestroy(stream);

    return;
}

void fzDecompress(uint8_t *deviceCompressed, float *deviceDecompressedOutput)
{
    // define the input info variables
    int inputSize = 0;
    int x = 0;
    int y = 0;
    int z = 0;
    float eb = 0;

    // copy the input information from the GPU global memory
    CHECK_CUDA(hipMemcpy(&inputSize, deviceCompressed, sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(&x, deviceCompressed + sizeof(int), sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(&y, deviceCompressed + 2 * sizeof(int), sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(&z, deviceCompressed + 3 * sizeof(int), sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(&eb, deviceCompressed + 4 * sizeof(int), sizeof(float), hipMemcpyDeviceToHost));

    uint8_t *deviceCompressedStartPosition;
    deviceCompressedStartPosition = deviceCompressed + sizeof(int) * 5;

    auto inputDimension = dim3(x, y, z);
    auto dataTypeLen = inputSize;

    float timeElapsed;

    uint16_t *deviceDecompressedQuantizationCode;

    uint16_t *deviceCompressedOutput;
    uint32_t *deviceBitFlagArr;
    uint32_t *deviceStartPosition;

    bool *deviceSignNum;

    std::chrono::time_point<std::chrono::system_clock> decompressionStart, decompressionEnd;

    int blockSize = 16;
    auto quantizationCodeByteLen = dataTypeLen * 2; // quantization code length in unit of bytes
    quantizationCodeByteLen = quantizationCodeByteLen % 4096 == 0 ? quantizationCodeByteLen : quantizationCodeByteLen - quantizationCodeByteLen % 4096 + 4096;
    auto paddingDataTypeLen = quantizationCodeByteLen / 2;
    int bitFlagArrSize = quantizationCodeByteLen % (blockSize * UINT32_BIT_LEN) == 0 ? quantizationCodeByteLen / (blockSize * UINT32_BIT_LEN) : int(quantizationCodeByteLen / (blockSize * UINT32_BIT_LEN)) + 1;

    dim3 block(32, 32);
    dim3 grid(floor(paddingDataTypeLen / 2048)); // divided by 2 is because the file is transformed from uint32 to uint16

    CHECK_CUDA(hipMalloc((void **)&deviceDecompressedQuantizationCode, sizeof(uint16_t) * paddingDataTypeLen));
    CHECK_CUDA(hipMalloc((void **)&deviceSignNum, sizeof(bool) * paddingDataTypeLen));

    // get the differents by the calculated offset
    int offsetCalculator = 0;
    deviceBitFlagArr = (uint32_t *)(deviceCompressedStartPosition + offsetCalculator);
    offsetCalculator += sizeof(uint32_t) * bitFlagArrSize;
    deviceStartPosition = (uint32_t *)(deviceCompressedStartPosition + offsetCalculator);
    offsetCalculator += sizeof(uint32_t) * floor(quantizationCodeByteLen / 4096);
    deviceCompressedOutput = (uint16_t *)(deviceCompressedStartPosition + offsetCalculator);

    hipStream_t stream;
    hipStreamCreate(&stream);

    decompressionStart = std::chrono::system_clock::now();

    // de-bitshuffle kernel
    decompressionFusedKernel<<<grid, block>>>((uint32_t *)deviceCompressedOutput, (uint32_t *)deviceDecompressedQuantizationCode, deviceBitFlagArr, deviceStartPosition);

    // de-pre-quantization
    cusz::experimental::launch_reconstruct_LorenzoI_var<float, uint16_t, float>(deviceSignNum, deviceDecompressedQuantizationCode, deviceDecompressedOutput, inputDimension, eb, timeElapsed, stream);

    hipDeviceSynchronize();
    decompressionEnd = std::chrono::system_clock::now();

    std::chrono::duration<double> decompressionTime = decompressionEnd - decompressionStart;

    std::cout << "decompression e2e time: " << decompressionTime.count() << " s\n";
    std::cout << "decompression e2e throughput: " << float(inputSize * sizeof(float)) / 1024 / 1024 / 1024 / decompressionTime.count() << " GB/s\n";

    CHECK_CUDA(hipFree(deviceDecompressedQuantizationCode));
    CHECK_CUDA(hipFree(deviceSignNum));

    hipStreamDestroy(stream);

    return;
}

extern "C"
{
    void pfzCompress(float **deviceInputArr,
                     int gpuIndex,
                     int deviceInputSize,
                     int deviceInputArrSize,
                     int worldSize,
                     float errorBound,
                     int *dimensionInfoArr,
                     uint8_t *deviceCompressed,
                     int *compressedSizeArr)
    {
        CHECK_CUDA(hipSetDevice(gpuIndex));
        int chunkSize = deviceInputSize / worldSize;
        int x, y, z;
        x = dimensionInfoArr[0];
        y = dimensionInfoArr[1];
        z = dimensionInfoArr[2];
        float eb = errorBound;
        int outputSizeCounter = 0;
        int outputSize = 0;
        uint16_t **quantizationCodeArr;
        quantizationCodeArr = (uint16_t **)malloc(sizeof(uint16_t *) * 4);

        for (int j = 0; j < worldSize; j++)
        {
            for (int i = 0; i < deviceInputArrSize; i++)
            {
                int actualInputSize = j == (worldSize - 1) ? deviceInputSize - chunkSize * (worldSize - 1) : chunkSize;
                std::cout << "actualInputSize: " << actualInputSize << std::endl;
                fzCompress(deviceInputArr[i] + chunkSize * j,
                           deviceCompressed + outputSizeCounter,
                           &outputSize,
                           actualInputSize,
                           x, y, z, eb,
                           quantizationCodeArr[j * 2 + i]);
                outputSizeCounter += outputSize;
                compressedSizeArr[j * deviceInputArrSize + i] = outputSize;
            }
        }
        uint16_t **hostQuantizationCodeArr;
        hostQuantizationCodeArr = (uint16_t **)malloc(sizeof(uint16_t *) * 8);
        for (int tmpIdx = 0; tmpIdx < 8; tmpIdx++)
        {
            hostQuantizationCodeArr[tmpIdx] = (uint16_t *)malloc(sizeof(uint16_t) * chunkSize);
            hipMemcpy(hostQuantizationCodeArr[tmpIdx],
                       quantizationCodeArr[tmpIdx],
                       sizeof(uint16_t) * chunkSize,
                       hipMemcpyDeviceToHost);
        }

        for (int tmpIdx = 0; tmpIdx < chunkSize; tmpIdx++)
        {
            bool res = false;
            res = (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[1][tmpIdx]) && (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[2][tmpIdx]) && (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[3][tmpIdx]) && (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[4][tmpIdx]) && (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[5][tmpIdx]) && (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[6][tmpIdx]) && (hostQuantizationCodeArr[0][tmpIdx] == hostQuantizationCodeArr[7][tmpIdx]);
            if (!res)
            {
                std::cout << "quantization code validation failed !!!!!" << std::endl
                          << "0: " << hostQuantizationCodeArr[0][tmpIdx] << std::endl
                          << "1: " << hostQuantizationCodeArr[1][tmpIdx] << std::endl
                          << "2: " << hostQuantizationCodeArr[2][tmpIdx] << std::endl
                          << "3: " << hostQuantizationCodeArr[3][tmpIdx] << std::endl
                          << "4: " << hostQuantizationCodeArr[4][tmpIdx] << std::endl
                          << "5: " << hostQuantizationCodeArr[5][tmpIdx] << std::endl
                          << "6: " << hostQuantizationCodeArr[6][tmpIdx] << std::endl
                          << "7: " << hostQuantizationCodeArr[7][tmpIdx] << std::endl
                          << "with index " << tmpIdx << std::endl
                          << "tmp index " << tmpIdx << std::endl;
                break;
            }
        }

        // validate input values
        float **hostInputArr;
        hostInputArr = (float **)malloc(sizeof(float *) * 8);
        for (int tmpIdx = 0; tmpIdx < 8; tmpIdx++)
        {
            hostInputArr[tmpIdx] = (float *)malloc(sizeof(float) * chunkSize);
            hipMemcpy(hostInputArr[tmpIdx],
                       deviceInputArr[tmpIdx / 4] + (chunkSize * tmpIdx % 4),
                       sizeof(float) * chunkSize,
                       hipMemcpyDeviceToHost);
        }
        for (int tmpIdx = 0; tmpIdx < chunkSize; tmpIdx++)
        {
            bool res = false;
            res = (hostInputArr[0][tmpIdx] == hostInputArr[1][tmpIdx]) && (hostInputArr[0][tmpIdx] == hostInputArr[2][tmpIdx]) && (hostInputArr[0][tmpIdx] == hostInputArr[3][tmpIdx]) && (hostInputArr[0][tmpIdx] == hostInputArr[4][tmpIdx]) && (hostInputArr[0][tmpIdx] == hostInputArr[5][tmpIdx]) && (hostInputArr[0][tmpIdx] == hostInputArr[6][tmpIdx]) && (hostInputArr[0][tmpIdx] == hostInputArr[7][tmpIdx]);
            if (!res)
            {
                std::cout << "input validation failed !!!!!" << std::endl
                          << "0: " << hostInputArr[0][tmpIdx] << std::endl
                          << "1: " << hostInputArr[1][tmpIdx] << std::endl
                          << "2: " << hostInputArr[2][tmpIdx] << std::endl
                          << "3: " << hostInputArr[3][tmpIdx] << std::endl
                          << "4: " << hostInputArr[4][tmpIdx] << std::endl
                          << "5: " << hostInputArr[5][tmpIdx] << std::endl
                          << "6: " << hostInputArr[6][tmpIdx] << std::endl
                          << "7: " << hostInputArr[7][tmpIdx] << std::endl
                          << "with index " << tmpIdx << std::endl;
                break;
            }
        }

        return;
    }

    void pfzDecompress(uint8_t *deviceCompressed,
                       int *offsetArr,
                       int offsetArrSize,
                       int gpuIndex,
                       float **deviceDecompressedOutput)
    {
        CHECK_CUDA(hipSetDevice(gpuIndex));
        for (int i = 0; i < offsetArrSize; i++)
        {
            fzDecompress(deviceCompressed + offsetArr[i],
                         deviceDecompressedOutput[i]);
        }

        return;
    }
}
